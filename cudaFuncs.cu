#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cudaFunction.cu
 Author      : Miriam Assraf
 Description : histogram calculation with CUDA
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "cudaFuncs.h"
#include "constants.h"

__global__ void histogramKernel(const int* A, int* results, int size, int num_blocks)
{
    int block = blockIdx.x;
    int thread = threadIdx.x;

    int start = (block * size) / num_blocks;
    int end = ((block + 1) * size) / num_blocks;

    for (int i = start; i < end; i++) {
        if (thread < HISTO_SZ) {
            if (A[i] == thread) {
                results[block * HISTO_SZ + thread]++;	// increase value thread (in range 0-255) for current block
            }
        }
    }
}

__host__ void checkErrors(hipError_t err, const char *error_msg)
{
	if (err != hipSuccess) {
			fprintf(stderr, error_msg, hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
}

int histogramWithCuda(const int* A, int* results, int size, int threadsPerBlock, int blocksPerGrid)
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls
	int* dev_A = 0;
    	int* dev_results = 0;

	// Allocate memory on GPU to copy the data from the host
   	err = hipMalloc((void**)&dev_A, size * sizeof(int));
    	checkErrors(err, "Failed to allocate device memory - %s\n");

	// Copy data from host to the GPU memory
	err = hipMemcpy(dev_A, A, size * sizeof(int), hipMemcpyHostToDevice);
	checkErrors(err, "Failed to copy data from host to device - %s\n");
	
	// Allocate memory on GPU for results to send to host
	err =hipMalloc((void**)&dev_results, blocksPerGrid* HISTO_SZ * sizeof(int));	// each block check on it's part for values 0-255
	checkErrors(err, "Failed to allocate device memory - %s\n");

	hipMemset(dev_results, 0, blocksPerGrid * HISTO_SZ * sizeof(int));	// initialize results array with zeros

	// Launch the Kernel
	histogramKernel << <blocksPerGrid, threadsPerBlock >> > (dev_A, dev_results, size, blocksPerGrid);
	err = hipGetLastError();
	checkErrors(err, "Failed to launch histogram kernel -  %s\n");

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(results, dev_results, blocksPerGrid*256 * sizeof(int), hipMemcpyDeviceToHost);
	checkErrors(err, "Failed to copy result array from device to host -%s\n");
	
	// Free allocated memory on GPU
	err = hipFree(dev_A);
	checkErrors(err, "Failed to free device data - %s\n");

	err = hipFree(dev_results);
	checkErrors(err, "Failed to free device results - %s\n");

	return 0;
}

